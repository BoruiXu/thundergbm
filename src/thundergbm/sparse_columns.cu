//
// Created by shijiashuai on 5/7/18.
//
#include <thundergbm/util/cub_wrapper.h>
#include <thundergbm/sparse_columns.h>

#include "thundergbm/sparse_columns.h"
#include "thundergbm/util/device_lambda.cuh"
#include "hipsparse.h"
#include "thundergbm/util/multi_device.h"

void SparseColumns::from_dataset(const DataSet &dataset) {
    LOG(INFO) << "copy csr matrix to GPU";
    //three arrays (on GPU/CPU) for csr representation
    SyncArray<float_type> val;
    SyncArray<int> col_idx;
    SyncArray<int> row_ptr;
    val.resize(dataset.csr_val.size());
    col_idx.resize(dataset.csr_col_idx.size());
    row_ptr.resize(dataset.csr_row_ptr.size());

    //copy data to the three arrays
    val.copy_from(dataset.csr_val.data(), val.size());
    col_idx.copy_from(dataset.csr_col_idx.data(), col_idx.size());
    row_ptr.copy_from(dataset.csr_row_ptr.data(), row_ptr.size());
    LOG(INFO) << "converting csr matrix to csc matrix";
    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    hipsparseCreate(&handle);
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);

    n_column = dataset.n_features_;
    nnz = dataset.csr_val.size();
    csc_val.resize(nnz);
    csc_row_idx.resize(nnz);
    csc_col_ptr.resize(n_column + 1);

    hipsparseScsr2csc(handle, dataset.n_instances(), n_column, nnz, val.device_data(), row_ptr.device_data(),
                     col_idx.device_data(), csc_val.device_data(), csc_row_idx.device_data(), csc_col_ptr.device_data(),
                     HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
    hipDeviceSynchronize();
    hipsparseDestroy(handle);
    hipsparseDestroyMatDescr(descr);
}


//FIXME remove this function
void correct_start(int *csc_col_ptr_2d_data, int first_col_start, int n_column_sub){
    device_loop(n_column_sub + 1, [=] __device__(int col_id) {
        csc_col_ptr_2d_data[col_id] = csc_col_ptr_2d_data[col_id] - first_col_start;
    });
};
void SparseColumns::to_multi_devices(vector<std::unique_ptr<SparseColumns>> &v_columns) const {
    //devide data into multiple devices
    int n_device = v_columns.size();
    int ave_n_columns = n_column / n_device;
    DO_ON_MULTI_DEVICES(n_device, [&](int device_id) {
        SparseColumns &columns = *v_columns[device_id];
        const int *csc_col_ptr_data = csc_col_ptr.host_data();
        int first_col_id = device_id * ave_n_columns;
        int n_column_sub = (device_id < n_device - 1) ? ave_n_columns : n_column - first_col_id;
        int first_col_start = csc_col_ptr_data[first_col_id];
        int nnz_sub = (device_id < n_device - 1) ?
                      (csc_col_ptr_data[(device_id + 1) * ave_n_columns] - first_col_start) : (nnz -
                                                                                               first_col_start);
        columns.column_offset = first_col_id + this->column_offset;
        columns.nnz = nnz_sub;
        columns.n_column = n_column_sub;
        columns.n_row = n_row;
        columns.csc_val.resize(nnz_sub);
        columns.csc_row_idx.resize(nnz_sub);
        columns.csc_col_ptr.resize(n_column_sub + 1);

        columns.csc_val.copy_from(csc_val.host_data() + first_col_start, nnz_sub);
        columns.csc_row_idx.copy_from(csc_row_idx.host_data() + first_col_start, nnz_sub);
        columns.csc_col_ptr.copy_from(csc_col_ptr.host_data() + first_col_id, n_column_sub + 1);

        int *csc_col_ptr_2d_data = columns.csc_col_ptr.device_data();
        correct_start(csc_col_ptr_2d_data, first_col_start, n_column_sub);
        //correct segment start positions
        LOG(TRACE) << "sorting feature values (multi-device)";
        cub_seg_sort_by_key(columns.csc_val, columns.csc_row_idx, columns.csc_col_ptr, false);
    });
    LOG(TRACE) << "sorting finished";
}

