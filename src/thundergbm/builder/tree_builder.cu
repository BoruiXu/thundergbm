//
// Created by jiashuai on 19-1-23.
//

#include <thundergbm/builder/tree_builder.h>
#include "thundergbm/util/multi_device.h"
#include "thundergbm/util/device_lambda.cuh"
#include <chrono>
#include <fstream>
typedef std::chrono::high_resolution_clock Clock;
#define TDEF(x_) std::chrono::high_resolution_clock::time_point x_##_t0, x_##_t1;
#define TSTART(x_) x_##_t0 = Clock::now();
#define TEND(x_) x_##_t1 = Clock::now();
#define TPRINT(x_, str) printf("%-20s \t%.6f\t sec\n", str, std::chrono::duration_cast<std::chrono::microseconds>(x_##_t1 - x_##_t0).count()/1e6);
#define TINT(x_) std::chrono::duration_cast<std::chrono::microseconds>(x_##_t1 - x_##_t0).count()

extern long long total_split_update_time;
void TreeBuilder::update_tree() {
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        auto& sp = this->sp[device_id];
        auto& tree = this->trees[device_id];
        auto sp_data = sp.device_data();
        LOG(DEBUG) << sp;
        int n_nodes_in_level = sp.size();

        Tree::TreeNode *nodes_data = tree.nodes.device_data();
        float_type rt_eps = param.rt_eps;
        float_type lambda = param.lambda;

        device_loop(n_nodes_in_level, [=]__device__(int i) {
            float_type best_split_gain = sp_data[i].gain;
            if (best_split_gain > rt_eps) {
                //do split
                if (sp_data[i].nid == -1) return;
                int nid = sp_data[i].nid;
                Tree::TreeNode &node = nodes_data[nid];
                node.gain = best_split_gain;

                Tree::TreeNode &lch = nodes_data[node.lch_index];//left child
                Tree::TreeNode &rch = nodes_data[node.rch_index];//right child
                lch.is_valid = true;
                rch.is_valid = true;
                node.split_feature_id = sp_data[i].split_fea_id;
                GHPair p_missing_gh = sp_data[i].fea_missing_gh;
                //todo process begin
                node.split_value = sp_data[i].fval;
                node.split_bid = sp_data[i].split_bid;

                lch.sum_gh_pair = sp_data[i].lch_sum_gh;
                if (!sp_data[i].default_right) {
                    lch.sum_gh_pair = lch.sum_gh_pair + p_missing_gh;
                    node.default_right = false;
                }
                else{
                    node.default_right = true;
                }
                rch.sum_gh_pair = node.sum_gh_pair - lch.sum_gh_pair;
                lch.calc_weight(lambda);
                rch.calc_weight(lambda);
            } else {
                //set leaf
                if (sp_data[i].nid == -1) return;
                int nid = sp_data[i].nid;
                Tree::TreeNode &node = nodes_data[nid];
                node.is_leaf = true;
                nodes_data[node.lch_index].is_valid = false;
                nodes_data[node.rch_index].is_valid = false;
            }
        });
        LOG(DEBUG) << tree.nodes;
    });
}

void TreeBuilder::predict_in_training(int k) {
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        auto y_predict_data = y_predict[device_id].device_data() + k * n_instances;
        auto nid_data = ins2node_id[device_id].device_data();
        const Tree::TreeNode *nodes_data = trees[device_id].nodes.device_data();
        auto lr = param.learning_rate;
        device_loop(n_instances, [=]__device__(int i) {
            int nid = nid_data[i];
            while (nid != -1 && (nodes_data[nid].is_pruned)) nid = nodes_data[nid].parent_index;
            y_predict_data[i] += lr * nodes_data[nid].base_weight;
        });
        //for(int i =0;i<5;i++){
        //    LOG(INFO)<<"in training prediction "<<i<<" "<<y_predict[device_id].host_data()[i];
        //}
    });
}

void TreeBuilder::init(const DataSet &dataset, const GBMParam &param) {
    int n_available_device;
    hipGetDeviceCount(&n_available_device);
    CHECK_GE(n_available_device, param.n_device) << "only " << n_available_device
                                                 << " GPUs available; please set correct number of GPUs to use";
    FunctionBuilder::init(dataset, param);
    this->n_instances = dataset.n_instances();
    trees = vector<Tree>(param.n_device);
    ins2node_id = MSyncArray<int>(param.n_device, n_instances);
    sp = MSyncArray<SplitPoint>(param.n_device);
    has_split = vector<bool>(param.n_device);
    int n_outputs = param.num_class * n_instances;
    if(param.num_class==2)
        n_outputs = n_instances;
    y_predict = MSyncArray<float_type>(param.n_device, n_outputs);
    gradients = MSyncArray<GHPair>(param.n_device, n_instances);
}

void TreeBuilder::ins2node_id_all_reduce(int depth) {
    //get global ins2node id
    {
        SyncArray<int> local_ins2node_id(n_instances);
        auto local_ins2node_id_data = local_ins2node_id.device_data();
        auto global_ins2node_id_data = ins2node_id.front().device_data();
        for (int d = 1; d < param.n_device; d++) {
            local_ins2node_id.copy_from(ins2node_id[d]);
            device_loop(n_instances, [=]__device__(int i) {
                global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
                                             global_ins2node_id_data[i] : local_ins2node_id_data[i];
            });
        }
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        ins2node_id[device_id].copy_from(ins2node_id.front());
    });
}

void TreeBuilder::split_point_all_reduce(int depth) {
    TIMED_FUNC(timerObj);
    //get global best split of each node
    int n_nodes_in_level = 1 << depth;//2^i
    int nid_offset = (1 << depth) - 1;//2^i - 1
    auto global_sp_data = sp.front().host_data();
    vector<bool> active_sp(n_nodes_in_level);

    for (int device_id = 0; device_id < param.n_device; device_id++) {
        auto local_sp_data = sp[device_id].host_data();
        for (int j = 0; j < sp[device_id].size(); j++) {
            int sp_nid = local_sp_data[j].nid;
            if (sp_nid == -1) continue;
            int global_pos = sp_nid - nid_offset;
            if (!active_sp[global_pos])
                global_sp_data[global_pos] = local_sp_data[j];
            else
                global_sp_data[global_pos] = (global_sp_data[global_pos].gain >= local_sp_data[j].gain)
                                             ?
                                             global_sp_data[global_pos] : local_sp_data[j];
            active_sp[global_pos] = true;
        }
    }
    //set inactive sp
    for (int n = 0; n < n_nodes_in_level; n++) {
        if (!active_sp[n])
            global_sp_data[n].nid = -1;
    }
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
       sp[device_id].copy_from(sp.front());
    });
    LOG(DEBUG) << "global best split point = " << sp.front();
}

vector<Tree> TreeBuilder::build_approximate(const MSyncArray<GHPair> &gradients) {
    vector<Tree> trees(param.tree_per_rounds);
    TIMED_FUNC(timerObj);
    DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
        this->shards[device_id].column_sampling(param.column_sampling_rate);
    });
    
    
    TDEF(split_update)
    for (int k = 0; k < param.tree_per_rounds; ++k) {
        Tree &tree = trees[k];
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            this->ins2node_id[device_id].resize(n_instances);
            this->gradients[device_id].set_device_data(const_cast<GHPair *>(gradients[device_id].device_data() + k * n_instances));
            this->trees[device_id].init2(this->gradients[device_id], param);
        });
        for (int level = 0; level < param.depth; ++level) {

            DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
                find_split(level, device_id);
            });

            split_point_all_reduce(level);
            {
                TIMED_SCOPE(timerObj, "apply sp");
                TSTART(split_update)
                update_tree();

                update_ins2node_id();
                TEND(split_update)
                total_split_update_time+=TINT(split_update);
                {
                    LOG(TRACE) << "gathering ins2node id";
                    //get final result of the reset instance id to node id
                    bool has_split = false;
                    for (int d = 0; d < param.n_device; d++) {
                        has_split |= this->has_split[d];
                    }
                    if (!has_split) {
                        LOG(INFO) << "no splittable nodes, stop";
                        break;
                    }
                }
                ins2node_id_all_reduce(level);
            }
        }
        DO_ON_MULTI_DEVICES(param.n_device, [&](int device_id){
            this->trees[device_id].prune_self(param.gamma);
        });
        predict_in_training(k);
        tree.nodes.resize(this->trees.front().nodes.size());
        tree.nodes.copy_from(this->trees.front().nodes);
    }
    
    //output nodes to files
    //std::ofstream outfile("node_info.txt");
    //int aa = 0;
    //for(int i =0;i<trees[0].nodes.size();i++){
    //    //LOG(INFO)<<"node "<<i<<" :";
    //    //LOG(INFO)<<trees[0].nodes.host_data()[i];
    //    //LOG(INFO)<<"*******************";
    //    auto tmp_node = trees[0].nodes.host_data();
    //    if(tmp_node[i].is_valid){
    //        outfile<<"tree node: "<<i<<", ";
    //        outfile<<"final node: "<<tmp_node[i].final_id<<", ";
    //        outfile<<"split feature: "<<tmp_node[i].split_feature_id<<", ";
    //        outfile<<"split value: "<<tmp_node[i].split_value<<", ";
    //        outfile<<"leaf node: "<<tmp_node[i].is_leaf<<", ";
    //        outfile<<"valid node: "<<!tmp_node[i].is_valid<<", ";
    //        outfile<<"node weight: "<<tmp_node[i].base_weight<<", ";
    //        outfile<<"node gain: "<<tmp_node[i].gain<<", ";
    //        outfile<<"node hess: "<<tmp_node[i].sum_gh_pair.h<<"\n";
    //    }

    //}
    //outfile.close();
    return trees;
}
